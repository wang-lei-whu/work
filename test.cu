#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <stdio.h>

using namespace std;
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//ld是维度，i是行，j是列，cuBLAS使用的是列存储
//这个宏表示， 矩阵第i行第j列的元素位置在C语言中 数组存储的索引
//ld表示 矩阵的第一个维的元素个数，就是 矩阵的行数。
__global__
void show(float* ptr, int size)
{
    for (int i = 0; i < size; i++)
        printf("%f\n", ptr);
}


void print_matrix(int R, int C, float* A, const char* name)
{
    printf("%s = \n", name);
    for (int r = 0; r < R; ++r)
    {
        printf("[");
        for (int c = 0; c < C; ++c)
        {
            printf("%10.6f", A[c * R + r]);
        }
        printf("] \n");
    }
}


void print_matrix_(int R, int C, float* A, const char* name)
{
    printf("%s = \n", name);
    for (int r = 0; r < R; ++r)
    {
        printf("[");
        for (int c = 0; c < C; ++c)
        {
            printf("%10.6f", A[r * C + c]);
        }
        printf("]\n");
    }
}

int main()
{
    int M = 6; //行数 矩阵A的行，结果矩阵C的行数.A=[3,9]
    int N = 4; //列数 矩阵A的列，矩阵B的列   B=[3,9]
    int B = 4; //行数 矩阵B的行
    int K = 3; //列数，结果矩阵C的列数  C=[3,3]

    //分配主机矩阵并初始化
    float* a, * b, * c;
    hipHostAlloc((void**)&a, sizeof(float) * M * N, hipHostMallocDefault);
    hipHostAlloc((void**)&b, sizeof(float) * B * N, hipHostMallocDefault);
    hipHostAlloc((void**)&c, sizeof(float) * M * K, hipHostMallocDefault);

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            a[IDX2C(i, j, M)] =  1.0;
        }
    }

    /*
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            a[i * N + j] = 1;
        }
    }

   
    print_matrix_(M, N, a, "A");

   
    */

    //可视化矩阵
    print_matrix(M, N, a, "A");

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < B; i++) {
            b[IDX2C(i, j, B)] = (float)(i * N + j + 1);
        }
    }

    /*
    for (int i = 0; i < B; ++i) {
        for (int j = 0; j < N; ++j) {
            b[i * N + j] = i * N + j + 1;
        }
    }
    */
    //print_matrix(B, N, b, "B");

    print_matrix(B, N, b, "B");
    //分配设备的数据
    float* d_a, * d_b, * d_c;
    hipMalloc(&d_a, sizeof(float) * M * N);
    hipMalloc(&d_b, sizeof(float) * B * N);
    hipMalloc(&d_c, sizeof(float) * M * K);

    //Host->device
    hipMemcpy(d_a, a, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * B * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeof(float) * M * K, hipMemcpyHostToDevice);
    //
    // print_matrix(B, N, d_b, "d_B");
    hipblasHandle_t handle;
    hipblasStatus_t ret;
    ret = hipblasCreate(&handle);

    //矩阵分块计算
    float* a_array[9], * b_array[9];
    float* c_array[9];

    int r = 3;
    int l = 3;

    for (int i = 0; i < r; ++i) {
        for (int j = 0; j < l; ++j) {
            a_array[i * l + j] = d_a + i * 9 + j * r;
            b_array[i * l + j] = d_b + i * 9 + j * r;
            c_array[i * l + j] = d_c + i * 9 + j * r;
            //printf("%d\n",*a_array[i * l + j]);
        }

    }
    //print_matrix_(r, l, *(a_array), "a_array");
    const float** d_Marray, ** d_Narray;
    float** d_Parray;
    hipMalloc((void**)&d_Marray, N * sizeof(float*));
    hipMalloc((void**)&d_Narray, N * sizeof(float*));
    hipMalloc((void**)&d_Parray, N * sizeof(float*));
    hipMemcpy(d_Marray, a_array, N * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_Narray, b_array, N * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_Parray, c_array, N * sizeof(float*), hipMemcpyHostToDevice);


    const float alpha = 1.0f;
    const float beta = 0.0f;

    //需要的是A矩阵的一个1行3列的矩阵乘以矩阵b的三行一列，
    int m =3; //按列 m = 1
    int n = 1; //按列 n = 3
    int k = 1; //按列 k = 1

    int lda = 9;
    int ldb = 9;
    int ldc = 9;
    int batch = 9;
    //    矩阵OP(Ａ)的维度是ｍ×ｋ
    //    矩阵OP(B)的维度是ｋ×ｎ
    //    矩阵C的维度是ｍ×ｎ
    //    运算为C = alpha * A * B + beta * C
    ret = hipblasSgemmBatched(handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        d_Narray, ldb,
        d_Marray, lda,
        &beta,
        d_Parray, ldc,
        batch);

    hipblasDestroy(handle);
   
    if (ret == HIPBLAS_STATUS_SUCCESS)
    {
        printf("sgemm success  %d, line(%d)\n", ret, __LINE__);
    }
   
    //show << <1, 1 >> > (c_array[0], 16);
    hipMemcpy(c, d_c, sizeof(float) * M * K, hipMemcpyDeviceToHost);

    print_matrix(M, K, c, "C = A x B");
    return 0;

}
